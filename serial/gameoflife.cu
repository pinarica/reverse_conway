#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <assert.h>
#include "png_util.h"
#define MAX_N 200 
#define CUDA_CALL(x) {hipError_t cuda_error__ = (x); if (cuda_error__) printf("CUDA error: " #x " returned \"%s\"\n", hipGetErrorString(cuda_error__));}

int n;
char* plate[2];
int which=0;

__global__ void cu_iteration(char * d_memblock,int n, int which)
{ 
   int threadn = blockDim.x * blockIdx.x + threadIdx.x;
   int i = threadn / n + 1;
   int j = threadn % n + 1; 
   int index = i * (n+2) + j; 

   if (i > n)
      return;
   if (j > n)
      return;

   char* cu_plate[2];
   cu_plate[0] = &d_memblock[0];
   cu_plate[1] = &d_memblock[(n+2)*(n+2)];

   int num = (cu_plate[which][index - n - 3]
        + cu_plate[which][index - n - 2]
        + cu_plate[which][index - n - 1]
        + cu_plate[which][index - 1]
        + cu_plate[which][index + 1]
        + cu_plate[which][index + n + 1]
        + cu_plate[which][index + n + 2]
        + cu_plate[which][index + n + 3]); 

   if(cu_plate[which][index]==1)
   {	
       cu_plate[!which][index] = (char) (num == 2 || num == 3) ?  1 : 0;
   } else {
       cu_plate[!which][index] = (char) (num == 3);
   }
}

void print_plate(){
    if (n < 60) {
        for(int i = 1; i <= n; i++){
            for(int j = 1; j <= n; j++){
                printf("%d", (int) plate[which][i * (n + 2) + j]);
            }
            printf("\n");
        }
    } else {
	printf("Plate too large to print to screen\n");
    }
    printf("\0");
}

void plate2png(const char filename[]) {
    unsigned char * img = (unsigned char *) malloc(n*n*sizeof(unsigned char));

    printf(filename);
    printf("\n");

    image_size_t sz;
    sz.width = n;
    sz.height = n; 

    for(int i = 1; i <= n; i++){
        for(int j = 1; j <= n; j++){
            int pindex = i * (n + 2) + j;
            int index = (i-1) * (n) + j;
            if (plate[!which][pindex] > 0)
		img[index] = (unsigned char) 255; 
            else 
		img[index] = (unsigned char) 0;
        }
    }
    write_png_file((char *) filename,img,sz);
    free(img);
    
}

int main() { 
    int M;
    struct hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, 0);
    printf("using %d multiprocessors\n",properties.multiProcessorCount);
    printf("max threads per processor: %d \n\n",properties.maxThreadsPerMultiProcessor);
    printf("max threads per block: %d \n\n",properties.maxThreadsPerBlock);
    n = 0;
    if(scanf("%d %d", &n, &M) == 2){
        int random=0;
        if (n == 0) { 
           n = MAX_N;
           random=1;
        }
        //Allocate memory for plates
        int arrlen = (n+2) * (n+2);
	int nBytes = sizeof(char)*arrlen;
        char *  memblock= (char *) malloc(nBytes*2);
        char *  d_memblock;
        plate[0] = (char *) &memblock[0];          
        plate[1] = (char *) &memblock[arrlen];          

        for(int k=0;k < 2*arrlen;k++)
		memblock[k] = (char) 0;

        char line[n];
        printf("Reading in %dx%d plate and running %d iterations\n",n,n,M);
        if (!random) {
            printf("reading plate in from standard input\n");
            for(int i = 1; i <= n; i++) {
                scanf("%s", &line);
                for(int j = 0; j < n; j++) {
                    plate[which][i * (n+2) + j + 1] = (char) line[j] - '0';
                }
            }
	} else {
            printf("generating random plate\n");
 	    for(int i = 1; i <= n; i++) 
                for(int j = 0; j < n; j++) { 
                   plate[which][i * (n+2) + j + 1] = rand() % 2;
                }
	}
        printf("Reading in %dx%d plate and running %d iterations\n",n,n,M);
	print_plate();

        int num_threads = min(properties.maxThreadsPerBlock,n*n);
        int num_blocks = ceil((double) n*n/ (double) num_threads);
        dim3 numThreads(num_threads,1,1);
        dim3 numBlocks(num_blocks,1,1);

        printf("totalCells=%d nBytes=%d num_threads=%d, num_blocks=%d\n",n*n, nBytes,num_threads,num_blocks);
	
	//CUDA Memory Copy
	CUDA_CALL(hipMalloc((void **) &d_memblock, nBytes*2));
	
   	printf("Copying to device..\n");
	CUDA_CALL(hipMemcpy(d_memblock, memblock, nBytes*2, hipMemcpyHostToDevice));
   	printf("Running Simulation...\n");
        for(int i = 0; i < M; i++){
            //CUDA Kernel Call
  	    printf("Iteration %d of %d\n",i,M);
            cu_iteration<<<numBlocks, numThreads>>>(d_memblock, n, which);
	    hipError_t errSync  = hipGetLastError();
	    hipError_t errAsync = hipDeviceSynchronize();
            if (errSync != hipSuccess) 
  		printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
            if (errAsync != hipSuccess)
  		printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
	    which=!which;
        }
	//CUDA Memory Copy
 	printf("Copying results to host..\n");   
	CUDA_CALL(hipMemcpy(memblock, d_memblock, nBytes*2, hipMemcpyDeviceToHost));
 
	plate2png("plate.png");
	print_plate();
    } else {
	printf("Input Format error on line 1\n");
    }
    return 0;
}
